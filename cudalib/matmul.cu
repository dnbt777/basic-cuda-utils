#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2048 // Size of the matrices (N x N)

__global__ void matmulKernel(float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0.0f;
        for (int k = 0; k < N; k++) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

extern "C" void matmul(float *A, float *B, float *C) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    // Allocate device memory and check for errors
    hipError_t err;
    err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory for A: %s\n", hipGetErrorString(err));
        return;
    }
    
    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory for B: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        return;
    }
    
    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory for C: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        return;
    }

    // Copy matrices from host to device
    err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying A to device: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying B to device: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // Launch the kernel
    matmulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
    
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
    }

    // Copy result from device to host
    err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying C to host: %s\n", hipGetErrorString(err));
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

extern "C" int cuda_device_check() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        printf("No CUDA-capable device found.\n");
        return -1; // Exit if no device is found
    }

    // Print device properties
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // Get properties of the first device
    printf("Using device: %s\n", deviceProp.name);

    return 0;
}

