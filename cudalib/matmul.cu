#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2 // Size of the matrices (N x N)

__global__ void matmulKernel(float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0;
        for (int k = 0; k < N; k++) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

extern "C" void matmul(float *A, float *B, float *C) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matmulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

extern "C" int cuda_device_check() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        printf("No CUDA-capable device found.\n");
        return -1; // Exit if no device is found
    }

    // Optionally, you can print device properties
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // Get properties of the first device
    printf("Using device: %s\n", deviceProp.name);

    // Example matrices
    float A[N * N] = {1, 2, 3, 4};
    float B[N * N] = {5, 6, 7, 8};
    float C[N * N] = {0};

    // Call the matrix multiplication function
    matmul(A, B, C);

    // Print the result
    printf("Result matrix C:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", C[i * N + j]);
        }
        printf("\n");
    }
    return 0;
}

